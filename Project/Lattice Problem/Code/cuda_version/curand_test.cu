

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>         // Input and output
#include <random>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h>
#include <assert.h>


__global__ void setup_kernel(hiprandState *state){
  int idx = threadIdx.x+blockDim.x*blockIdx.x;
  hiprand_init(0, idx, 0, &state[idx]);
}

__global__ void generateAll(double *result, hiprandState *state){

  int idx = threadIdx.x + blockDim.x*blockIdx.x;
  result[idx] = hiprand_uniform_double(&state[idx]);
}

__global__ void generateSingle(double* N, hiprandState *state, int i){

  int idx = threadIdx.x + blockDim.x*blockIdx.x;
  if (i == idx) {
    *N = hiprand_uniform_double(&state[idx]);
  }
}

int main(){
  int BlockSize = 10;

  hiprandState *d_state1, *d_state2;
  hipMalloc((void**)&d_state1, BlockSize*sizeof(hiprandState));
  hipMalloc((void**)&d_state2, BlockSize*sizeof(hiprandState));

  setup_kernel<<<1,BlockSize>>>(d_state1);
  setup_kernel<<<1,BlockSize>>>(d_state2);

  double *d_result;
  double *h_result = new double[BlockSize];
  hipMalloc((void**)&d_result,  BlockSize*sizeof(double));


  generateAll<<<1,BlockSize>>>(d_result, d_state1);
  hipMemcpy(h_result,  d_result,  BlockSize*sizeof(double), hipMemcpyDeviceToHost);


  std::cout << "Generating all at once:" << std::endl;
  for(int i = 0; i < BlockSize; i++){
    std::cout << h_result[i] << ", ";
  }
  std::cout << std::endl;


  std::cout << "Generating one at a time:" << std::endl;
  double* d_N;
  double* N = new double;
  hipMalloc((void**)&d_N,sizeof(double));


  for(int i = 0; i < BlockSize; i++){
    generateSingle<<<1,BlockSize>>>(d_N, d_state2,i);
    hipMemcpy(N, d_N, sizeof(double), hipMemcpyDeviceToHost);

    // std::cout << *N << ", ";
  }
  std::cout << std::endl;

  return 0;
}
