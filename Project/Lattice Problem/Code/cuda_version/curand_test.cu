

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h>
#include <assert.h>
// #define ITER 10

__global__ void setup_kernel(hiprandState *state){

  int idx = threadIdx.x+blockDim.x*blockIdx.x;
  hiprand_init(1234, idx, 0, &state[idx]);
}

__global__ void generate_kernel(hiprandState *my_curandstate, float *result, int *resultp){
    int idx = threadIdx.x + blockDim.x*blockIdx.x;
    result[idx] = hiprand_uniform(&my_curandstate[idx])+1;
    resultp[idx] = hiprand_poisson(&my_curandstate[idx], 0);
}

int main(){
  int ITER = 10;

  hiprandState *d_state;
  hipMalloc(&d_state, ITER*sizeof(hiprandState));

  float *d_result;
  int *d_resultp;
  float *h_result = (float*) malloc(ITER*sizeof(float));
  int *h_resultp = (int*) malloc(ITER*sizeof(int));
  hipMalloc(&d_result, ITER*sizeof(float));
  hipMalloc(&d_resultp, ITER*sizeof(int));
  setup_kernel<<<1,ITER>>>(d_state);

  generate_kernel<<<1,ITER>>>(d_state, d_result, d_resultp);
  hipMemcpy(h_result, d_result, ITER*sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(h_resultp, d_resultp, ITER*sizeof(int), hipMemcpyDeviceToHost);
  for(int i = 0; i < ITER; i++){
    printf("result : %f \n" , h_result[i]);
    printf("resultp: %d \n" , h_resultp[i]);
  }

  return 0;
}