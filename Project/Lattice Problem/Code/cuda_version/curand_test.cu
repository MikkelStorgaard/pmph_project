

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <random>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h>
#include <assert.h>
// #define ITER 10

__device__ int RandP(hiprandState rng_state, double lambda) {

  double L = exp(-lambda);
  double p = 1.0;
  int k = 0;
  while (p > L) {
    k++;
    double u = hiprand_uniform_double(&rng_state);
    p *= u;
  }
  return k - 1;

}


__global__ void setup_kernel(hiprandState *state){

  int idx = threadIdx.x+blockDim.x*blockIdx.x;
  hiprand_init(1234, idx, 0, &state[idx]);
}

__global__ void generate_kernel(hiprandState *my_curandstate, int *result, int *resultp){
    int idx = threadIdx.x + blockDim.x*blockIdx.x;
    result[idx] = RandP(my_curandstate[idx],0.1);
    resultp[idx] = hiprand_poisson(&my_curandstate[idx], 0.1);
}

int main(){
  int ITER = 1000;

  hiprandState *d_state;
  hipMalloc(&d_state, ITER*sizeof(hiprandState));

  int *d_result;
  int *d_resultp;
  int *h_result  = (int*) malloc(ITER*sizeof(int));
  int *h_resultp = (int*) malloc(ITER*sizeof(int));
  hipMalloc(&d_result,  ITER*sizeof(int));
  hipMalloc(&d_resultp, ITER*sizeof(int));
  setup_kernel<<<1,ITER>>>(d_state);

  generate_kernel<<<1,ITER>>>(d_state, d_result, d_resultp);
  hipMemcpy(h_result,  d_result,  ITER*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(h_resultp, d_resultp, ITER*sizeof(int), hipMemcpyDeviceToHost);

  // Set limit on distribution
  std::mt19937 rng;
  std::poisson_distribution <long long> distr(0.1);


  printf("\n\nRandP:\n");
  for(int i = 0; i < ITER; i++){
    if (h_result[i] > 0) {
     printf("%d, ",h_result[i]);
    }
  }
  printf("\n\nstd library:\n");


  for(int i = 0; i < ITER; i++){
    int k = (int)distr(rng);
    if (k>0) {
     printf("%d, ",k);
    }
  }
  printf("\n");




  return 0;
}